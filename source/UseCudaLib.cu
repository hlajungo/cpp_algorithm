#include "UseCudaLib.h"

bool Use_cuda::get_has_gpu()
{
  return has_gpu;
}

Use_cuda::Use_cuda() : has_gpu(false)
{
  if (is_has_gpu())
  {
    set_has_gpu(true);
  }
}

void Use_cuda::set_has_gpu(bool val)
{
  has_gpu = val;
}

bool Use_cuda::is_has_gpu()
{
  int device_count = 0;
  hipError_t err = hipGetDeviceCount(&device_count);

  if (err != hipSuccess)
  {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    return false;
  }

  return device_count > 0;
}
